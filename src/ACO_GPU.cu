#include "hip/hip_runtime.h"
#include "ACO_GPU.hpp"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <limits>
#include <iostream>
#include <stdexcept>
#include <cassert>

#define MAX_VERTICES 64
#define MAX_ANTS     1024
#define INF_WEIGHT   1e6f  

__global__ void setup_kernel(hiprandState *state, unsigned long seed, int n) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
        hiprand_init(seed, id, 0, &state[id]);
}

__global__ void aco_kernel(
    int numVertices,
    int numAnts,
    const float* graph,         // n x n
    float* pheromones,          // n x n
    float alpha,
    float beta,
    float Q,
    int* outPaths,              // numAnts x (numVertices+1)
    float* outLengths,
    hiprandState* states)
{
    int antId = blockIdx.x * blockDim.x + threadIdx.x;
    if (antId >= numAnts) return;

    int visited[MAX_VERTICES] = {0};
    int path[MAX_VERTICES + 1];

    hiprandState localState = states[antId];
    int start = hiprand(&localState) % numVertices;
    int current = start;
    path[0] = current;
    visited[current] = 1;
    float length = 0.0f;

    for (int step = 1; step < numVertices; ++step) {
        float probs[MAX_VERTICES] = {0};
        float sumProb = 0.0f;
        for (int j = 0; j < numVertices; ++j) {
            if (!visited[j]) {
                float weight = graph[current * numVertices + j];
                if (weight < 1e-5f)
                    continue;
                float tau = powf(pheromones[current * numVertices + j], alpha);
                float eta = powf(1.0f / weight, beta);
                probs[j] = tau * eta;
                sumProb += probs[j];
            }
        }
        float pick = hiprand_uniform(&localState) * sumProb;
        float acc = 0.0f;
        int next = -1;
        for (int j = 0; j < numVertices; ++j) {
            if (!visited[j] && probs[j] > 0) {
                acc += probs[j];
                if (acc >= pick) {
                    next = j;
                    break;
                }
            }
        }
        if (next == -1) {
            for (int j = 0; j < numVertices; ++j)
                if (!visited[j] && graph[current * numVertices + j] > 1e-5f) {
                    next = j; break;
                }
        }
        if (next == -1) {
            for (int fill = step; fill <= numVertices; ++fill) path[fill] = current;
            length += INF_WEIGHT * (numVertices - step + 1);
            break;
        }

        path[step] = next;
        visited[next] = 1;
        length += graph[current * numVertices + next];
        current = next;
    }
    float retWeight = graph[current * numVertices + start];
    if (retWeight > 1e-5f) {
        path[numVertices] = start;
        length += retWeight;
    } else {
        path[numVertices] = current;
        length += INF_WEIGHT;
    }


    for (int i = 0; i <= numVertices; ++i)
        outPaths[antId * (numVertices + 1) + i] = path[i];
    outLengths[antId] = length;
    states[antId] = localState;
}

inline void CUDA_ASSERT(hipError_t code, const char* file, int line, const char* msg = "") {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << " - " << hipGetErrorString(code) << " " << msg << std::endl;
        throw std::runtime_error("CUDA error");
    }
}
#define CUDA_ASSERT(val) CUDA_ASSERT((val), __FILE__, __LINE__)


ACO_GPU::ACO_GPU(const Graph& graph, int numAnts, float alpha, float beta, float evaporation, float Q)
    : numAnts(numAnts), alpha(alpha), beta(beta), evaporation(evaporation), Q(Q), bestLength(std::numeric_limits<float>::max())
{
    numVertices = graph.size();
    assert(numVertices <= MAX_VERTICES && "Zwiększ MAX_VERTICES!");
    assert(numAnts <= MAX_ANTS && "Zwiększ MAX_ANTS!");

    // Konwertuj double -> float (CUDA)
    const std::vector<double>& src = graph.data();
    hostGraphMatrix.resize(numVertices * numVertices);
    for (int i = 0; i < numVertices * numVertices; ++i)
        hostGraphMatrix[i] = static_cast<float>(src[i]);
    bestPath.clear();
}

void ACO_GPU::run(int iterations) {
    float* d_graph = nullptr;
    float* d_pheromones = nullptr;
    int* d_paths = nullptr;
    float* d_lengths = nullptr;
    hiprandState* d_states = nullptr;

    size_t matSize = numVertices * numVertices * sizeof(float);
    size_t pathSize = numAnts * (numVertices + 1) * sizeof(int);
    size_t lenSize = numAnts * sizeof(float);

    CUDA_ASSERT(hipMalloc(&d_graph, matSize));
    CUDA_ASSERT(hipMalloc(&d_pheromones, matSize));
    CUDA_ASSERT(hipMalloc(&d_paths, pathSize));
    CUDA_ASSERT(hipMalloc(&d_lengths, lenSize));
    CUDA_ASSERT(hipMalloc(&d_states, numAnts * sizeof(hiprandState)));

    CUDA_ASSERT(hipMemcpy(d_graph, hostGraphMatrix.data(), matSize, hipMemcpyHostToDevice));
    std::vector<float> hostPheromones(numVertices * numVertices, 1.0f);
    CUDA_ASSERT(hipMemcpy(d_pheromones, hostPheromones.data(), matSize, hipMemcpyHostToDevice));

    setup_kernel<<<(numAnts+127)/128, 128>>>(d_states, 12345, numAnts);
    hipDeviceSynchronize();

    std::vector<int> h_paths(numAnts * (numVertices + 1));
    std::vector<float> h_lengths(numAnts);

    for (int iter = 0; iter < iterations; ++iter) {
        aco_kernel<<<(numAnts+127)/128, 128>>>(numVertices, numAnts, d_graph, d_pheromones,
                                               alpha, beta, Q, d_paths, d_lengths, d_states);
        hipDeviceSynchronize();

        CUDA_ASSERT(hipMemcpy(h_paths.data(), d_paths, pathSize, hipMemcpyDeviceToHost));
        CUDA_ASSERT(hipMemcpy(h_lengths.data(), d_lengths, lenSize, hipMemcpyDeviceToHost));

        for (int i = 0; i < numAnts; ++i) {
            if (h_lengths[i] < bestLength) {
                bestLength = h_lengths[i];
                bestPath.assign(h_paths.begin() + i*(numVertices+1), h_paths.begin() + (i+1)*(numVertices+1));
            }
        }

        // Aktualizacja feromonów na CPU
        for (auto& p : hostPheromones) p *= (1.0f - evaporation);
        for (int i = 0; i < numAnts; ++i) {
            float len = h_lengths[i];
            if (len < INF_WEIGHT * 0.9f) { // ignoruj niepełne cykle!
                float delta = Q / (len > 0.001f ? len : 0.001f);
                for (int j = 0; j < numVertices; ++j) {
                    int from = h_paths[i*(numVertices+1) + j];
                    int to   = h_paths[i*(numVertices+1) + j + 1];
                    hostPheromones[from * numVertices + to] += delta;
                    hostPheromones[to   * numVertices + from] += delta; // symetria
                }
            }
        }
        for (auto& p : hostPheromones) if (p < 0.0001f) p = 0.0001f;

        CUDA_ASSERT(hipMemcpy(d_pheromones, hostPheromones.data(), matSize, hipMemcpyHostToDevice));
    }

    hipFree(d_graph); hipFree(d_pheromones); hipFree(d_paths); hipFree(d_lengths); hipFree(d_states);
}

const std::vector<int>& ACO_GPU::getBestPath() const {
    return bestPath;
}

float ACO_GPU::getBestPathLength() const {
    return bestLength;
}
